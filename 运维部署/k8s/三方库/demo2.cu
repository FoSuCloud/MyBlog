#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h> // 引入这个头文件，用来调用sleep函数

// 这是一个在GPU上运行的函数，叫做kernel
__global__ void add(int *a, int *b, int *c)
{
    // 计算线程的全局索引
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int r = rand() % 10 + 1;
    // 把结果存储到c[id]指向的内存中
    c[id] = a[id] * r + b[id] / r;
}

int main()
{
    // 定义三个整数数组的大小
    int N = 10240;
    // 定义三个整数数组变量
    int a[N];
    int b[N];
    int c[N];

    // 定义三个指针变量，用来指向GPU上的内存
    int *dev_a;
    int *dev_b;
    int *dev_c;

    // 在GPU上分配三块内存，大小为N个整数，把地址赋给dev_a, dev_b, dev_c
    hipMalloc(&dev_a, N * sizeof(int));
    hipMalloc(&dev_b, N * sizeof(int));
    hipMalloc(&dev_c, N * sizeof(int));

    // 定义一个无限循环
    while (1)
    {
        // 用随机数初始化a和b数组
        for (int i = 0; i < N; i++)
        {
            a[i] = rand() % 100;
            b[i] = rand() % 100;
        }

        // 把CPU上的a和b数组中的数据复制到GPU上的内存中
        hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

        // 调用add函数，在GPU上执行，传入dev_a, dev_b, dev_c作为参数
        // 使用320个线程和320个块，总共10240个线程
        add<<<320, 320>>>(dev_a, dev_b, dev_c);

        // 把GPU上的内存中的数据复制到CPU上的c数组中
        hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

        // 打印结果
        for (int i = 0; i < N; i++)
        {
            printf("%d + %d = %d\n", a[i], b[i], c[i]);
        }

        // 等待一秒
        sleep(1);
    }

    // 释放GPU上的内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
