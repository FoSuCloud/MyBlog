
#include <hip/hip_runtime.h>
#include <stdio.h>

// 这是一个在GPU上运行的函数，叫做kernel
__global__ void add(int a, int b, int *c)
{
    // 计算a + b，并把结果存储到c指向的内存中
    *c = a + b;
}

int main()
{
    // 定义三个整数变量
    int a = 2;
    int b = 7;
    int c;

    // 定义一个指针变量，用来指向GPU上的内存
    int *dev_c;

    // 在GPU上分配一块内存，大小为一个整数，把地址赋给dev_c
    hipMalloc(&dev_c, sizeof(int));

    // 调用add函数，在GPU上执行，传入a, b, dev_c作为参数
    add<<<1, 1>>>(a, b, dev_c);

    // 把GPU上的内存中的数据复制到CPU上的变量c中
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    // 打印结果
    printf("%d + %d = %d\n", a, b, c);

    // 释放GPU上的内存
    hipFree(dev_c);

    return 0;
}
